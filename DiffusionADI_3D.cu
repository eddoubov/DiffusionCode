#include "hip/hip_runtime.h"
// Import libraries
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <magma_v2.h>
#include <magma_types.h>
#include <magma_lapack.h>
#include <math.h>

const int numStreams = 1;
const int maxThreadsPerBlock = 1024;
const int TILE_DIM = 32;
const int BLOCK_ROWS = 8;

// Store variables into constant memory
__device__ __constant__ int dev_N;
__device__ __constant__ double dev_lambda;
__device__ __constant__ double dev_phi_C;
__device__ __constant__ double dev_alpha;
__device__ __constant__ double dev_beta;

//
__global__ void test_func();
//__global__ void init_grid(double* u, int update);
//__global__ void init_A(double* A);
__global__ void expl_x(double* u, double* du, double dt);
__global__ void expl_y(double* u, double* du, double dt);
__global__ void expl_z(double* u, double* du, double dt);
__global__ void transpose(double* u, double* uT, int dir);
__global__ void comb_u(double* u1, double* u2, double* u3, int pom);
__global__ void add_react_term(double* u, double* uN, int mod_bool);
__global__ void copy(double* u1, double* u2);

void swap (int *a, int *b) {
  int temp = *a;
  //printf("%d\n", *a);
  *a = *b;
  *b = temp;
}

void randomize (int array[], int n, long int seed) {
  
  for (int i = n-1; i > 0; i--) {
    int j = lrand48() % (i+1);

    if (i == 10) {
      printf("%d\n", j);
    }

    swap(&array[i], &array[j]);
  }
}

void get_coords (int index, int localN, int* x_coord, int* y_coord, int* z_coord) {

  int a = index;
  *z_coord = a/(localN*localN);
  int b = a % (localN*localN);
  *y_coord = b/(localN);
  int c = b % localN;
  *x_coord = c;
  
}


int main(int argc, char* argv[]) {

  // Read in inputs
  const int N = atol(argv[1]);
  double dx = atof(argv[2]);
  double tau = atof(argv[3]);
  double D = atof(argv[4]);
  double R = atof(argv[5]);
  double C = atof(argv[6]);
  double phi_C = atof(argv[7]);
  double cap_vf = atof(argv[8]);
  int num_iter = atol(argv[9]);

  long int seed;
  if (argc > 10) {
    seed = atol(argv[100]);
  } else {

    FILE* urand = fopen("/dev/urandom", "r");

    fread( &seed, sizeof(long int), 1, urand);

    fclose(urand);
  }

  int localN = N/numStreams;
  int gridsize = localN*localN*localN;

  long int Vt = N*N*N;
  long int Nc = Vt*cap_vf;
  double mu = N/cbrt(Nc);
  int sigma = 60/dx;

  printf("%ld\n", Nc);
  printf("%lf\n", mu);

  // Define grid and thread dimensions
  // Turn the following into an if statement later
  // to account for N > 1024
  //const int tPB2D = localN;
  //const int bPG2D = localN;

  // Define variable for ADI scheme
  double lambda = (tau*D)/(pow(dx,2));
  double time_step = tau;
  double alpha = time_step*R;
  double beta = time_step*C;
  printf("%lf\n", lambda);
  printf("%lf\n", alpha);

  // Start Monte-Carlo Method
  srand48(seed);
  printf("%ld\n", seed);
  
  int *rand_indices, *cap_indices;
  hipHostAlloc((void**)&rand_indices, gridsize*sizeof(int), hipHostMallocDefault);
  hipHostAlloc((void**)&cap_indices, gridsize*sizeof(int), hipHostMallocDefault);

  for (int i = 0; i < gridsize; ++i) {
    rand_indices[i] = i;
  }

  printf("%d\n", rand_indices[10]);
  
  randomize(rand_indices, gridsize, seed);

  printf("%d\n", rand_indices[10]);
  
  int cap_count = 0;
  int x_coord, y_coord, z_coord;
  int x_cap, y_cap, z_cap;
  int temp_index, cap_index;
  
  for (int i=0; i< gridsize; ++i) {

    temp_index = rand_indices[i];
    
    get_coords(temp_index, localN, &x_coord, &y_coord, &z_coord);
    
    if (i == 0) {
      cap_indices[cap_count] = temp_index;
      cap_count++;
      continue;
    }

    double min_dist_cap;
    
    for (int j=0; j<cap_count; ++j) {
      cap_index = cap_indices[j];
      get_coords(cap_index, localN, &x_cap, &y_cap, &z_cap);

      double expr = (x_coord-x_cap)*(x_coord-x_cap) + (y_coord-y_cap)*(y_coord-y_cap)
	+ (z_coord-z_cap)*(z_coord-z_cap);
      double dist = sqrt(expr);

      if (j==0) {
	min_dist_cap = dist;
      } else if(dist < min_dist_cap) {
	min_dist_cap = dist;
      }
    }    

    if (i == 10) {
      printf("%d,", x_cap);
      printf("%d,", y_cap);
      printf("%d\n", z_cap);
    }
    
  }
  
  
  magma_init();
  magma_int_t *piv, info;
  magma_int_t m = localN;
  magma_int_t n = localN;
  magma_int_t err;
  
  //Declare matrices on host
  double *u, *du, *uT, *uN, *A;
  //double *u1, *u2, *u3, *u4, *u5, *u6, *u7, *u8, *u9, *u10;

  // Declare matrices for device
  double* dev_u[numStreams];
  double* dev_du[numStreams];
  double* dev_uT[numStreams];
  double* dev_uN[numStreams];
  magmaDouble_ptr dev_A;

  /**
  double* dev_u1[numStreams];
  double* dev_u2[numStreams];
  double* dev_u3[numStreams];
  double* dev_u4[numStreams];
  double* dev_u5[numStreams];
  double* dev_u6[numStreams];
  double* dev_u7[numStreams];
  double* dev_u8[numStreams];
  double* dev_u9[numStreams];
  double* dev_u10[numStreams];
  **/
  
  // Send varibales to constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(dev_N), &N, sizeof(const int));
  hipMemcpyToSymbol(HIP_SYMBOL(dev_lambda), &lambda, sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(dev_phi_C), &phi_C, sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(dev_alpha), &alpha, sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(dev_beta), &beta, sizeof(double));

  // Initialize memory on host
  hipHostAlloc((void**)&u, gridsize*sizeof(double), hipHostMallocDefault);
  hipHostAlloc((void**)&du, gridsize*sizeof(double), hipHostMallocDefault);
  hipHostAlloc((void**)&uT, gridsize*sizeof(double), hipHostMallocDefault);  //delete later
  hipHostAlloc((void**)&uN, gridsize*sizeof(double), hipHostMallocDefault);  //delete later

  /**
  hipHostAlloc((void**)&u1, localN*localN*localN*sizeof(double), hipHostMallocDefault);
  hipHostAlloc((void**)&u2, localN*localN*localN*sizeof(double), hipHostMallocDefault);
  hipHostAlloc((void**)&u3, localN*localN*localN*sizeof(double), hipHostMallocDefault);
  hipHostAlloc((void**)&u4, localN*localN*localN*sizeof(double), hipHostMallocDefault);
  hipHostAlloc((void**)&u5, localN*localN*localN*sizeof(double), hipHostMallocDefault);
  hipHostAlloc((void**)&u6, localN*localN*localN*sizeof(double), hipHostMallocDefault);
  hipHostAlloc((void**)&u7, localN*localN*localN*sizeof(double), hipHostMallocDefault);
  hipHostAlloc((void**)&u8, localN*localN*localN*sizeof(double), hipHostMallocDefault);
  hipHostAlloc((void**)&u9, localN*localN*localN*sizeof(double), hipHostMallocDefault);
  hipHostAlloc((void**)&u10, localN*localN*localN*sizeof(double), hipHostMallocDefault);
  **/
  
  err = magma_dmalloc_cpu(&A, m*m);

  if (err) {
    printf("Issue in memory allocation cpu: A\n");
    exit(1);
  }

  // Initialize memory on device
  hipStream_t stream[numStreams];
  for (int i=0; i<numStreams; ++i) {
    hipMalloc((void**)&dev_u[i], gridsize*sizeof(double));
    hipMalloc((void**)&dev_du[i], gridsize*sizeof(double));
    hipMalloc((void**)&dev_uT[i], gridsize*sizeof(double));
    hipMalloc((void**)&dev_uN[i], gridsize*sizeof(double));

    /**
    hipMalloc((void**)&dev_u1[i], localN*localN*localN*sizeof(double));
    hipMalloc((void**)&dev_u2[i], localN*localN*localN*sizeof(double));
    hipMalloc((void**)&dev_u3[i], localN*localN*localN*sizeof(double));
    hipMalloc((void**)&dev_u4[i], localN*localN*localN*sizeof(double));
    hipMalloc((void**)&dev_u5[i], localN*localN*localN*sizeof(double));
    hipMalloc((void**)&dev_u6[i], localN*localN*localN*sizeof(double));
    hipMalloc((void**)&dev_u7[i], localN*localN*localN*sizeof(double));
    hipMalloc((void**)&dev_u8[i], localN*localN*localN*sizeof(double));
    hipMalloc((void**)&dev_u9[i], localN*localN*localN*sizeof(double));
    hipMalloc((void**)&dev_u10[i], localN*localN*localN*sizeof(double));
    **/
    
    err = magma_dmalloc(&dev_A, m*m);
    if (err) {
      printf("Issue in memory allocation gpu\n");
      exit(1);
    }
    
    hipStreamCreate( &(stream[i]) );
  }

  piv = (magma_int_t*)malloc(m*sizeof(magma_int_t));

  int ind;
  
  for (int j=0; j<localN; ++j) {
    for (int i=0; i<localN; ++i) {
      ind = i + j*localN;
      if ((i == 0 && j == 0) || (i == localN-1 && j == localN-1)) {
	A[ind] = 1;
      } else if ((i == 1 && j == 0) || (i == (localN-2) && j == (localN-1))) {
	A[ind] = 0;
      } else if (i-j == 0) {
	A[ind] = 1 + lambda;
      } else if (i-j == -1 || i-j == 1) {
	A[ind] = -lambda/2;
      } else {
	A[ind] = 0;
      }
    }
  }

  magma_dsetmatrix(m, m, A, m, dev_A, m, 0);

  int update = 0;
  
  for (int k=0; k<localN; ++k) {
    for (int j=0; j<localN; ++j) {
      for (int i=0; i<localN; ++i) {
	ind  = i + j*localN + k*localN*localN;

	if (i == 0 || j == 0 || k == 0) {
	  //printf("left_corner\n");
	  u[ind] = phi_C;
	} else if (i == localN-1 || j == localN-1 || k == localN-1) {
	  //printf("right_corner\n");
	  u[ind] = phi_C;
	} else if (update == 0) {
	  u[ind] = phi_C;
	}
      }
    }
  }
  
  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);  

  dim3 bPG3D(localN,localN);
  //dim3 block(threadsPerBlock, threadsPerBlock, threadsPerBlock);
  dim3 tPB3D(localN);
  dim3 bPG2D_trans(localN/TILE_DIM, localN/TILE_DIM, localN);
  dim3 tPB2D_trans(TILE_DIM, BLOCK_ROWS, 1);
  
  for (int i=0; i<numStreams; ++i) {
    // Initialize grid using kernel
    //init_grid<<<bPG3D, tPB3D, 0, stream[i]>>>(dev_u[i], 0);
    // Initialize implicit matrix
    //init_A<<<bPG2D, tPB2D, 0, stream[i]>>>(dev_A[i]);

    magma_dgetmatrix(m, n, dev_A, m, A+i*localN, m, 0);
    magma_dgetrf_gpu(m, m, dev_A, m, piv, &info); 
    
    for (int j=0; j<num_iter; ++j) {
      // Iterate explicitly in the x direction using kernel
      //test_func<<<grid, block>>>();
      hipMemcpyAsync(dev_u[i], u+i*localN, gridsize*sizeof(double), hipMemcpyHostToDevice, stream[i]);

      if (j > 0) {
	hipMemcpyAsync(dev_du[i], du+i*localN, gridsize*sizeof(double), hipMemcpyHostToDevice, stream[i]);
	hipMemcpyAsync(dev_uN[i], uN+i*localN, gridsize*sizeof(double), hipMemcpyHostToDevice, stream[i]);
      }
      
      add_react_term<<<bPG3D, tPB3D, 0, stream[i]>>>(dev_u[i], dev_uN[i], 0);
      comb_u<<<bPG2D_trans, tPB2D_trans, 0, stream[i]>>>(dev_u[i], dev_uN[i], dev_uN[i], 1);
      //copy<<<bPG2D_trans, tPB2D_trans, 0, stream[i]>>>(dev_u[i], dev_u1[i]);  // copy for debugging
      expl_x<<<bPG3D, tPB3D, 0, stream[i]>>>(dev_u[i], dev_du[i], 2);
      comb_u<<<bPG2D_trans, tPB2D_trans, 0, stream[i]>>>(dev_uN[i], dev_du[i], dev_uN[i], 1);
      //copy<<<bPG2D_trans, tPB2D_trans, 0, stream[i]>>>(dev_uN[i], dev_u1[i]);  // copy for debugging
      expl_y<<<bPG3D, tPB3D, 0, stream[i]>>>(dev_u[i], dev_du[i], 1);
      comb_u<<<bPG2D_trans, tPB2D_trans, 0, stream[i]>>>(dev_uN[i], dev_du[i], dev_uN[i], 1);
      //copy<<<bPG2D_trans, tPB2D_trans, 0, stream[i]>>>(dev_uN[i], dev_u2[i]);  //copy for debugging
      expl_z<<<bPG3D, tPB3D, 0, stream[i]>>>(dev_u[i], dev_du[i], 1);
      comb_u<<<bPG2D_trans, tPB2D_trans, 0, stream[i]>>>(dev_uN[i], dev_du[i], dev_uN[i], 1);
      //copy<<<bPG2D_trans, tPB2D_trans, 0, stream[i]>>>(dev_uN[i], dev_u3[i]);  //copy for debugging
      magma_dgetmatrix(m, n, dev_A, m, A+i*localN, m, 0);
      magma_dgetrs_gpu(MagmaTrans, m, n*n, dev_A, m, piv, dev_uN[i], m, &info);
      //copy<<<bPG2D_trans, tPB2D_trans, 0, stream[i]>>>(dev_uN[i], dev_u4[i]);  //copy for debugging
      add_react_term<<<bPG3D, tPB3D, 0, stream[i]>>>(dev_uN[i], dev_uT[i], 0);
      comb_u<<<bPG2D_trans, tPB2D_trans, 0, stream[i]>>>(dev_uN[i], dev_uT[i], dev_uN[i], 1);
      expl_y<<<bPG3D, tPB3D, 0, stream[i]>>>(dev_u[i], dev_du[i], 2);
      comb_u<<<bPG2D_trans, tPB2D_trans, 0, stream[i]>>>(dev_uN[i], dev_du[i], dev_uN[i], 0);
      //copy<<<bPG2D_trans, tPB2D_trans, 0, stream[i]>>>(dev_uN[i], dev_u5[i]);  //copy for debugging
      transpose<<<bPG2D_trans, tPB2D_trans, 0, stream[i]>>>(dev_uN[i], dev_uT[i], 1);
      //copy<<<bPG2D_trans, tPB2D_trans, 0, stream[i]>>>(dev_uT[i], dev_u6[i]);  //copy for debugging
      magma_dgetmatrix(m, n, dev_A, m, A+i*localN, m, 0);
      magma_dgetrs_gpu(MagmaTrans, m, n*n, dev_A, m, piv, dev_uT[i], m, &info);
      //copy<<<bPG2D_trans, tPB2D_trans, 0, stream[i]>>>(dev_uT[i], dev_u7[i]);  //copy for debugging
      transpose<<<bPG2D_trans, tPB2D_trans, 0, stream[i]>>>(dev_uT[i], dev_uN[i], 1);
      add_react_term<<<bPG3D, tPB3D, 0, stream[i]>>>(dev_uN[i], dev_uT[i], 0);
      comb_u<<<bPG2D_trans, tPB2D_trans, 0, stream[i]>>>(dev_uN[i], dev_uT[i], dev_uN[i], 1);
      expl_z<<<bPG3D, tPB3D, 0, stream[i]>>>(dev_u[i], dev_du[i], 2);
      comb_u<<<bPG2D_trans, tPB2D_trans, 0, stream[i]>>>(dev_uN[i], dev_du[i], dev_uN[i], 0);
      //copy<<<bPG2D_trans, tPB2D_trans, 0, stream[i]>>>(dev_uN[i], dev_u8[i]);  //copy for debugging
      transpose<<<bPG2D_trans, tPB2D_trans, 0, stream[i]>>>(dev_uN[i], dev_uT[i], 0);
      magma_dgetmatrix(m, n, dev_A, m, A+i*localN, m, 0);
      //copy<<<bPG2D_trans, tPB2D_trans, 0, stream[i]>>>(dev_uT[i], dev_u9[i]);  //copy for debugging
      magma_dgetrs_gpu(MagmaTrans, m, n*n, dev_A, m, piv, dev_uT[i], m, &info);
      transpose<<<bPG2D_trans, tPB2D_trans, 0, stream[i]>>>(dev_uT[i], dev_u[i], 0);
      //copy<<<bPG2D_trans, tPB2D_trans, 0, stream[i]>>>(dev_u[i], dev_u10[i]);  //copy for debugging
      //init_grid<<<bPG3D, tPB3D, 0, stream[i]>>>(dev_u[i], 1);
      hipMemcpyAsync(u+i*localN, dev_u[i], gridsize*sizeof(double), hipMemcpyDeviceToHost, stream[i]);
      hipMemcpyAsync(du+i*localN, dev_du[i], gridsize*sizeof(double), hipMemcpyDeviceToHost, stream[i]);
      hipMemcpyAsync(uN+i*localN, dev_uN[i], gridsize*sizeof(double), hipMemcpyDeviceToHost, stream[i]);
      // Transpose grid in kernel
      //transpose<<<blocksPerGrid, threadsPerBlock, 0, stream[i]>>>(dev_du[i], dev_uT[i]);
      // Iterate implicitly in the y direction in kernel
      //magma_dgetrs_gpu(MagmaTrans, m, n, dev_A[i], m, piv, dev_uT[i], m, &info);
      //expl_x<<<blocksPerGrid, threadsPerBlock, 0, stream[i]>>>(dev_uT[i], dev_du[i]);
      //transpose<<<blocksPerGrid, threadsPerBlock, 0, stream[i]>>>(dev_du[i], dev_u[i]);
      //magma_dgetrs_gpu(MagmaTrans, m, n, dev_A[i], m, piv, dev_u[i], m, &info);
    }
  }
  
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);

  
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);

  /**
  for (int i=0; i<numStreams; ++i) {
    //hipMemcpyAsync(du+i*localN, dev_du[i], localN*localN*localN*sizeof(double), hipMemcpyDeviceToHost, stream[i]);
    //hipMemcpyAsync(u+i*localN, dev_u[i], localN*localN*localN*sizeof(double), hipMemcpyDeviceToHost, stream[i]);
    hipMemcpyAsync(u1+i*localN, dev_u1[i], localN*localN*localN*sizeof(double), hipMemcpyDeviceToHost, stream[i]);
    hipMemcpyAsync(u2+i*localN, dev_u2[i], localN*localN*localN*sizeof(double), hipMemcpyDeviceToHost, stream[i]);
    hipMemcpyAsync(u3+i*localN, dev_u3[i], localN*localN*localN*sizeof(double), hipMemcpyDeviceToHost, stream[i]);
    hipMemcpyAsync(u4+i*localN, dev_u4[i], localN*localN*localN*sizeof(double), hipMemcpyDeviceToHost, stream[i]);
    hipMemcpyAsync(u5+i*localN, dev_u5[i], localN*localN*localN*sizeof(double), hipMemcpyDeviceToHost, stream[i]);
    hipMemcpyAsync(u6+i*localN, dev_u6[i], localN*localN*localN*sizeof(double), hipMemcpyDeviceToHost, stream[i]);
    hipMemcpyAsync(u7+i*localN, dev_u7[i], localN*localN*localN*sizeof(double), hipMemcpyDeviceToHost, stream[i]);
    hipMemcpyAsync(u8+i*localN, dev_u8[i], localN*localN*localN*sizeof(double), hipMemcpyDeviceToHost, stream[i]);
    hipMemcpyAsync(u9+i*localN, dev_u9[i], localN*localN*localN*sizeof(double), hipMemcpyDeviceToHost, stream[i]);
    hipMemcpyAsync(u10+i*localN, dev_u10[i], localN*localN*localN*sizeof(double), hipMemcpyDeviceToHost, stream[i]);
  }
  **/

  for (int i=0; i<numStreams; ++i)
    hipStreamSynchronize(stream[i]);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  
  for (int i = 0; i<numStreams; ++i) {
    hipFree(dev_u[i]);
    hipFree(dev_du[i]);
    hipFree(dev_uT[i]);
    hipFree(dev_uN[i]);

    /**
    hipFree(dev_u1[i]);
    hipFree(dev_u2[i]);
    hipFree(dev_u3[i]);
    hipFree(dev_u4[i]);
    hipFree(dev_u5[i]);
    hipFree(dev_u6[i]);
    hipFree(dev_u7[i]);
    hipFree(dev_u8[i]);
    hipFree(dev_u9[i]);
    hipFree(dev_u10[i]);
    **/
  }

  //printf("cuda_test: %lf\n", u1[17]);
  
  for (int i = 0; i<numStreams; ++i)
    hipStreamDestroy(stream[i]);

  //dgetri(m, A, n, piv
  
  char fn1[25], fn2[23];
  //char fn3[25], fn4[25], fn5[25], fn6[25], fn7[25], fn8[25], fn9[25], fn10[25], fn11[25], fn12[27];

  for (int j=0; j<N; ++j) {

    int offset = j*N*N;

    sprintf(fn1, "du_files/Test_du_%d.csv", j);
    sprintf(fn2, "u_files/Test_u_%d.csv", j);
    /**
    sprintf(fn3, "u1_files/Test_u1_%d.csv", j);
    sprintf(fn4, "u2_files/Test_u2_%d.csv", j);
    sprintf(fn5, "u3_files/Test_u3_%d.csv", j);
    sprintf(fn6, "u4_files/Test_u4_%d.csv", j);
    sprintf(fn7, "u5_files/Test_u5_%d.csv", j);
    sprintf(fn8, "u6_files/Test_u6_%d.csv", j);
    sprintf(fn9, "u7_files/Test_u7_%d.csv", j);
    sprintf(fn10, "u8_files/Test_u8_%d.csv", j);
    sprintf(fn11, "u9_files/Test_u9_%d.csv", j);
    sprintf(fn12, "u10_files/Test_u10_%d.csv", j);
    **/
    
    FILE *fileid = fopen(fn1, "w");

    for (int i=0; i<(N*N); ++i) {
      if (i % N == 0) {
	fprintf(fileid, "%lf", du[i+offset]);
      } else if (i % N < (N-1)) {
	fprintf(fileid, ",%lf", du[i+offset]);
      } else {
	fprintf(fileid, ",%lf\n", du[i+offset]);
      }
    }

    fclose(fileid);
  
    FILE *fileid2 = fopen(fn2, "w");
  
    for (int i=0; i<(N*N); ++i) {
      //printf("%lf\n", u[i+offset]);
      //printf("%d\n", offset);
      if (i % N == 0) {
	fprintf(fileid2, "%lf", u[i+offset]);
      } else if (i % N < (N-1)) {
	fprintf(fileid2, ",%lf", u[i+offset]);
      } else {
	fprintf(fileid2, ",%lf\n", u[i+offset]);
      }
    }

    fclose(fileid2);

    /**
    FILE *fileid3 = fopen(fn3, "w");
  
    for (int i=0; i<(N*N); ++i) {
      //printf("%d\n", i);
      //printf("%lf\n", u1[i+offset]);
      if (i % N == 0) {
	fprintf(fileid3, "%lf", u1[i+offset]);
      } else if (i % N < (N-1)) {
	fprintf(fileid3, ",%lf", u1[i+offset]);
      } else {
	fprintf(fileid3, ",%lf\n", u1[i+offset]);
      }
    }

    fclose(fileid3);

    
    FILE *fileid4 = fopen(fn4, "w");
  
    for (int i=0; i<(N*N); ++i) {
      if (i % N == 0) {
	fprintf(fileid4, "%lf", u2[i+offset]);
      } else if (i % N < (N-1)) {
	fprintf(fileid4, ",%lf", u2[i+offset]);
      } else {
	fprintf(fileid4, ",%lf\n", u2[i+offset]);
      }
    }

    fclose(fileid4);

    FILE *fileid5 = fopen(fn5, "w");
  
    for (int i=0; i<(N*N); ++i) {
      if (i % N == 0) {
	fprintf(fileid5, "%lf", u3[i+offset]);
      } else if (i % N < (N-1)) {
	fprintf(fileid5, ",%lf", u3[i+offset]);
      } else {
	fprintf(fileid5, ",%lf\n", u3[i+offset]);
      }
    }

    fclose(fileid5);

    FILE *fileid6 = fopen(fn6, "w");
  
    for (int i=0; i<(N*N); ++i) {
      if (i % N == 0) {
	fprintf(fileid6, "%lf", u4[i+offset]);
      } else if (i % N < (N-1)) {
	fprintf(fileid6, ",%lf", u4[i+offset]);
      } else {
	fprintf(fileid6, ",%lf\n", u4[i+offset]);
      }
    }

    fclose(fileid6);

    FILE *fileid7 = fopen(fn7, "w");
  
    for (int i=0; i<(N*N); ++i) {
      if (i % N == 0) {
	fprintf(fileid7, "%lf", u5[i+offset]);
      } else if (i % N < (N-1)) {
	fprintf(fileid7, ",%lf", u5[i+offset]);
      } else {
	fprintf(fileid7, ",%lf\n", u5[i+offset]);
      }
    }

    fclose(fileid7);

    FILE *fileid8 = fopen(fn8, "w");
  
    for (int i=0; i<(N*N); ++i) {
      if (i % N == 0) {
	fprintf(fileid8, "%lf", u6[i+offset]);
      } else if (i % N < (N-1)) {
	fprintf(fileid8, ",%lf", u6[i+offset]);
      } else {
	fprintf(fileid8, ",%lf\n", u6[i+offset]);
      }
    }

    fclose(fileid8);

    FILE *fileid9 = fopen(fn9, "w");
  
    for (int i=0; i<(N*N); ++i) {
      if (i % N == 0) {
	fprintf(fileid9, "%lf", u7[i+offset]);
      } else if (i % N < (N-1)) {
	fprintf(fileid9, ",%lf", u7[i+offset]);
      } else {
	fprintf(fileid9, ",%lf\n", u7[i+offset]);
      }
    }

    fclose(fileid9);

    FILE *fileid10 = fopen(fn10, "w");
  
    for (int i=0; i<(N*N); ++i) {
      if (i % N == 0) {
	fprintf(fileid10, "%lf", u8[i+offset]);
      } else if (i % N < (N-1)) {
	fprintf(fileid10, ",%lf", u8[i+offset]);
      } else {
	fprintf(fileid10, ",%lf\n", u8[i+offset]);
      }
    }

    fclose(fileid10);

    FILE *fileid11 = fopen(fn11, "w");
  
    for (int i=0; i<(N*N); ++i) {
      if (i % N == 0) {
	fprintf(fileid11, "%lf", u9[i+offset]);
      } else if (i % N < (N-1)) {
	fprintf(fileid11, ",%lf", u9[i+offset]);
      } else {
	fprintf(fileid11, ",%lf\n", u9[i+offset]);
      }
    }

    fclose(fileid11);

    FILE *fileid12 = fopen(fn12, "w");
  
    for (int i=0; i<(N*N); ++i) {
      if (i % N == 0) {
	fprintf(fileid12, "%lf", u10[i+offset]);
	} else if (i % N < (N-1)) {
	fprintf(fileid12, ",%lf", u10[i+offset]);
	} else {
	fprintf(fileid12, ",%lf\n", u10[i+offset]);
	}
      }

    fclose(fileid12);
    **/
  }

  FILE *fileid13 = fopen("Test_A.csv", "w");
  
  for (int i=0; i<(N*N); ++i)
    if (i % N == 0) {
      fprintf(fileid13, "%lf", A[i]);
    } else if (i % N < (N-1)) {
      fprintf(fileid13, ",%lf", A[i]);
    } else {
      fprintf(fileid13, ",%lf\n", A[i]);
    }

  fclose(fileid13);
    
  free(A);
  magma_free(dev_A);
  free(piv);
  magma_finalize();
  
  hipHostFree(u);
  hipHostFree(du);
  hipHostFree(uN);
  hipHostFree(uT);

  /**
  hipHostFree(u1);
  hipHostFree(u2);
  hipHostFree(u3);
  hipHostFree(u4);
  hipHostFree(u5);
  hipHostFree(u6);
  hipHostFree(u7);
  hipHostFree(u8);
  hipHostFree(u9);
  hipHostFree(u10);
  **/
  
  printf("Kernel Time: %gms\n", elapsedTime);
  
  return 0;
}

  /**
__global__ void test_func() {
  
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  int k = threadIdx.z + blockIdx.z * blockDim.z;
  printf("GPU - i = %d, j = %d, k = %d\n", i, j, k);
  printf("Threadx = %d, Blockx = %d, Thready = %d, Blocky = %d\n", threadIdx.x, blockIdx.x, threadIdx.y, blockIdx.y);
  
}
  **/

/**
__global__ void init_grid(double* u, int update) {
  int l_i = threadIdx.x;
  int l_j = blockIdx.x;
  int l_k = blockIdx.y;

  int g_i = l_i + dev_N*l_j + dev_N*dev_N*l_k;
  
  if (l_i == 0 || l_j == 0 || l_k == 0) {
    //printf("left_corner\n");
    u[g_i] = dev_phi_C;
  } else if (l_i == dev_N-1 || l_j == dev_N-1 || l_k == dev_N-1) {
    //printf("right_corner\n");
    u[g_i] = dev_phi_C;
  } else if (update == 0) {
    u[g_i] = dev_phi_C;
  }
}
**/

/**
__global__ void init_A(double* A) {
  int l_i = threadIdx.x;
  int l_j = blockIdx.x;

  int g_i = l_i + dev_N*l_j;
  
  if ((l_i == 0 && l_j == 0) || (l_i == dev_N-1 && l_j == dev_N-1)) {
    A[g_i] = 1;
  } else if ((l_i == 1 && l_j == 0) || (l_i == (dev_N-2) && l_j == (dev_N-1))) {
    A[g_i] = 0;
  } else if (l_i-l_j == 0) {
    A[g_i] = 1 + dev_lambda;
  } else if (l_i-l_j == -1 || l_i-l_j == 1) {
    A[g_i] = -dev_lambda/2;
  } else {
    A[g_i] = 0;
  }
}
**/

__global__ void expl_x(double* u, double* du, double dt) {
  __shared__ double localu[maxThreadsPerBlock];
  int l_i = threadIdx.x;
  int l_j = blockIdx.x;
  int l_k = blockIdx.y;

  // Index to pull data from global into local
  int g_i = l_i + dev_N*l_j + dev_N*dev_N*l_k;
  
  localu[l_i] = u[g_i];
  
  __syncthreads();

  double temp_lambda = dev_lambda/dt;
  
  if (l_i == 0 || l_i == (dev_N - 1)) {
    du[g_i] = 0;
  } else if (l_j == 0 || l_j == (dev_N - 1)) {
    du[g_i] = 0;
  } else if (l_k == 0 || l_k == (dev_N - 1)) {
    du[g_i] = 0;
  } else {
    du[g_i] = (temp_lambda*localu[l_i+1] - 2*temp_lambda*localu[l_i] + temp_lambda*localu[l_i-1]);
  }
}


__global__ void expl_y(double* u, double* du, double dt) {
  __shared__ double localu[maxThreadsPerBlock];

  int l_i = threadIdx.x;
  int l_j = blockIdx.x;
  int l_k = blockIdx.y;

  // Index to pull data from global into local
  int g_i = l_j + dev_N*l_i + dev_N*dev_N*l_k;
  
  localu[l_i] = u[g_i];
  
  __syncthreads();

  double temp_lambda = dev_lambda/dt;
  
  if (l_i == 0 || l_i == (dev_N - 1)) {
    du[g_i] = 0;
  } else if (l_j == 0 || l_j == (dev_N - 1)) {
    du[g_i] = 0;
  } else if (l_k == 0 || l_k == (dev_N - 1)) {
    du[g_i] = 0;
  } else {
    du[g_i] = (temp_lambda*localu[l_i+1] - 2*temp_lambda*localu[l_i] + temp_lambda*localu[l_i-1]);
  }
}

__global__ void expl_z(double* u, double* du, double dt) {
  __shared__ double localu[maxThreadsPerBlock];

  int l_i = threadIdx.x;
  int l_j = blockIdx.x;
  int l_k = blockIdx.y;

  // Index to pull data from global into local
  int g_i = l_k + dev_N*l_j + dev_N*dev_N*l_i;
  
  localu[l_i] = u[g_i];
  
  __syncthreads();

  double temp_lambda = dev_lambda/dt;
  
  if (l_i == 0 || l_i == (dev_N - 1)) {
    du[g_i] = 0;
  } else if (l_j == 0 || l_j == (dev_N - 1)) {
    du[g_i] = 0;
  } else if (l_k == 0 || l_k == (dev_N - 1)) {
    du[g_i] = 0;
  } else {
    du[g_i] = (temp_lambda*localu[l_i+1] - 2*temp_lambda*localu[l_i] + temp_lambda*localu[l_i-1]);
  }
  
}

__global__ void transpose(double* idata, double* odata, int dir) {
  __shared__ double localu[TILE_DIM][TILE_DIM];

  if (dir == 1) {

    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int z = blockIdx.z * dev_N * dev_N;
    int width = gridDim.x * TILE_DIM;

    //printf("%d\n", gridDim.x);
    
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
      
      localu[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x + z];
      
    }
    
    __syncthreads();

    x = blockIdx.y * TILE_DIM + threadIdx.x;
    y = blockIdx.x * TILE_DIM + threadIdx.y;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
      
      odata[(y+j)*width + x + z] = localu[threadIdx.x][threadIdx.y + j];
      
    }

  } else if (dir == 0) {

    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int z = blockIdx.y * TILE_DIM + threadIdx.y;
    int y = blockIdx.z * dev_N;
    int width = gridDim.x * TILE_DIM * dev_N;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
      
      localu[threadIdx.y+j][threadIdx.x] = idata[(z+j)*width + x + y];
      
    }
    
    __syncthreads();

    x = blockIdx.y * TILE_DIM + threadIdx.x;
    z = blockIdx.x * TILE_DIM + threadIdx.y;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
      
      odata[(z+j)*width + x + y] = localu[threadIdx.x][threadIdx.y + j];
      
    }
  }
}

__global__ void comb_u(double* mat1, double* mat2, double* mat3, int pom) {
  /**
  int l_i = threadIdx.x;
  int l_j = blockIdx.x;
  int l_k = blockIdx.y;

  int g_i = l_i + dev_N*l_j + dev_N*dev_N*l_k;
  **/
  
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int z = blockIdx.z * dev_N * dev_N;
  int width = gridDim.x * TILE_DIM;
  
  if (pom == 1) {
    //mat3[g_i] = mat1[g_i] + mat2[g_i];
    for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
      mat3[(y+j)*width + x + z] = mat1[(y+j)*width + x + z] + mat2[(y+j)*width + x + z];
  } else {
    //mat3[g_i] = mat1[g_i] - mat2[g_i];
    for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
      mat3[(y+j)*width + x + z] = mat1[(y+j)*width + x + z] - mat2[(y+j)*width + x + z];
  }
}

__global__ void add_react_term(double* idata, double* odata, int mod_bool) {
  int l_i = threadIdx.x;
  int l_j = blockIdx.x;
  int l_k = blockIdx.y;

  int g_i = l_i + dev_N*l_j + dev_N*dev_N*l_k;

  double react_term;
  double phi_V = dev_phi_C - 10;
  
  if (mod_bool == 1) {
    if (idata[g_i] > phi_V) {
      react_term = -dev_alpha*idata[g_i]/2;
    } else {
      react_term = 0;
    }
  } else {
    if (idata[g_i] > phi_V) {
      react_term = -dev_alpha*idata[g_i]/2 - dev_beta/2;
    } else if (idata[g_i] < phi_V && idata[g_i] > dev_beta) {
      react_term = -dev_beta/2;
    } else {
      react_term = -idata[g_i];
    }
  }

  if (l_i == 0 || l_i == (dev_N - 1)) {
    odata[g_i] = 0;
  } else if (l_j == 0 || l_j == (dev_N - 1)) {
    odata[g_i] = 0;
  } else if (l_k == 0 || l_k == (dev_N - 1)) {
    odata[g_i] = 0;
  } else {
    odata[g_i] = react_term;
  }
  
}

__global__ void copy(double* cdata1, double* cdata2) {
  //int l_i = threadIdx.x;
  //int l_j = blockIdx.x;
  //int l_k = blockIdx.y;

  //int g_i = l_i + dev_N*l_j + dev_N*dev_N*l_k;
  
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int z = blockIdx.z * dev_N * dev_N;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
    cdata2[(y+j)*width + x + z] = cdata1[(y+j)*width + x + z];
  
  /**
  if (l_k == 0 && l_j == 0 && l_i == 0) {
    printf("%lf\n", u1[g_i]);
  }
  **/
  
  //odata[g_i] = idata[g_i]; 
}

